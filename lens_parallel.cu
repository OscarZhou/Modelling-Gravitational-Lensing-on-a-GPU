#include "hip/hip_runtime.h"
/* CUDA timing example

   To compile: nvcc -o testprog2 testprog2.cu

 */
#include <iostream>
#include <string>
#include <cmath>

#include <hip/hip_runtime.h>
#include "lenses.h"
#include "arrayff.hxx"


// Global variables! Not nice style, but we'll get away with it here.

// Boundaries in physical units on the lens plane
const float WL  = 2.0;
const float XL1 = -WL;
const float XL2 =  WL;
const float YL1 = -WL;
const float YL2 =  WL;


// Kernel that executes on the CUDA device. This is executed by ONE
// stream processor
__global__ void cudaShoot(float* device_array, int npixx, float* xlens, float* ylens, float* eps,
        float rsrc2, float xsrc, float ysrc, int nlenses, int N)
{
  // What element of the array does this thread work on
    float xl, yl, xs, ys, sep2, mu;

    const float ldc  = 0.5;      // limb darkening coefficient

    const float lens_scale = 0.005;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < N)
    {

        yl = YL1 + (tid / npixx) * lens_scale;
        xl = XL1 + (tid % npixx) * lens_scale;

        float xd, yd;
        int numuse = 0;
        float dx, dy, dr;

        xs = xl;
        ys = yl;
        for (int p = 0; p < nlenses; ++p) {
          dx = xl - xlens[p];
          dy = yl - ylens[p];
          dr = dx * dx + dy * dy;
          xs -= eps[p] * dx / dr;
          ys -= eps[p] * dy / dr;
        }


         xd = xs -xsrc;
         yd = ys -ysrc;
         sep2 = xd * xd + yd * yd;
         if (sep2 < rsrc2)
         {
             mu = sqrt (1 - sep2 / rsrc2);
             device_array[tid] = 1.0 - ldc * (1 - mu);
         }
         else
         {
             device_array[tid] = 0;
         }

    }

}

// main routine that executes on the host
int main(void)
{
    float* xlens;
    float* ylens;
    float* eps;
    const int nlenses = set_example_n(100, &xlens, &ylens, &eps);
    size_t lensize = nlenses*sizeof(int);
    std::cout << "# Simulating " << nlenses << " lens system" << std::endl; //nelnses = 1

    const float rsrc = 0.1;      // radius
    const float ldc  = 0.5;      // limb darkening coefficient
    const float xsrc = 0.0;      // x and y centre on the map
    const float ysrc = 0.0;

    const float lens_scale = 0.005;

    const int npixx = static_cast<int>(floor((XL2 - XL1) / lens_scale)) + 1;
    const int npixy = static_cast<int>(floor((YL2 - YL1) / lens_scale)) + 1;   //npixx = 801, npixy = 801
    std::cout << "# Building " << npixx << "X" << npixy << " lens image" << std::endl;


    int npitotal = npixx * npixy;
    size_t size = npitotal * sizeof(float);
    // CUDA event types used for timing execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate in HOST memory
    float* host_plane = (float*)malloc(size);


    // Initialize vectors
    for (int i = 0; i < npitotal; ++i) {
        host_plane[i] = 0;
    }


    // Allocate in DEVICE memory
    float *device_plane, *dev_xlens, *dev_ylens, *dev_eps;
    hipMalloc(&device_plane, size);
    hipMalloc(&dev_xlens, lensize);
    hipMalloc(&dev_ylens, lensize);
    hipMalloc(&dev_eps, lensize);



    // Copy vectors from host to device memory
    //hipMemcpy(device_plane, host_plane, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_xlens, xlens, lensize, hipMemcpyHostToDevice);
    hipMemcpy(dev_ylens, ylens, lensize, hipMemcpyHostToDevice);
    hipMemcpy(dev_eps, eps, lensize, hipMemcpyHostToDevice);



    // Set up layout of kernel grid
    int threadsPerBlock = 1024;
    int blocksPerGrid = (npitotal + threadsPerBlock - 1) / threadsPerBlock;


    // Put the lens image in this array
    Array<float, 2> lensim(npixy, npixx);

    const float rsrc2 = rsrc * rsrc;

    // Launch kernel and time it
    hipEventRecord(start, 0);


    cudaShoot<<<blocksPerGrid, threadsPerBlock>>>(device_plane, npixx, dev_xlens, dev_ylens, dev_eps, rsrc2, xsrc, ysrc, nlenses, npitotal);


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);




  float time;  // Must be a float
  hipEventElapsedTime(&time, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  std::cout << "Kernel took: " << time << " ms" << std::endl;

  // Copy result from device memory into host memory
  hipMemcpy(host_plane, device_plane, size, hipMemcpyDeviceToHost);

  for (int i=0; i< npixy; i++)
      for(int j =0 ; j< npixx; j++)
      {
          lensim(i, j) = host_plane[i*npixx + j];
      }

  // Free device memory
  hipFree(device_plane);
  hipFree(dev_xlens);
  hipFree(dev_ylens);
  hipFree(dev_eps);

  dump_array<float, 2>(lensim, "lens3.fit");

  delete[] xlens;
  delete[] ylens;
  delete[] eps;

  free(host_plane);
}
